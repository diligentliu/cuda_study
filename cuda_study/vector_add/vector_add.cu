#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <vector>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <glog/logging.h>

#include "cuda_study/util/util.h"
#include "cuda_study/util/perf_util.h"

// CUDA kernel：每个线程计算一个元素
__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        C[idx] = A[idx] + B[idx];
}

int main() {
    int N = 1 << 20;
    size_t size = N * sizeof(float);

    std::vector<float> h_A(N);
    std::vector<float> h_B(N);
    std::vector<float> h_C(N);

    // 初始化输入数据
    std::mt19937 gen(std::random_device{}());
    std::normal_distribution<float> dist(0.0f, 0.2f);
    for (int i = 0; i < N; i++) {
        h_A[i] = dist(gen);
        h_B[i] = dist(gen);
    }

    // host time cost test
    std::vector<float> h_C_host(N);
    auto host_test_func = [&]() {
        for (int i = 0; i < N; i++) {
            h_C_host[i] = h_A[i] + h_B[i];
        }
    };
    float avg_time_host, throughput_host;
    util::perf_single_threaded(host_test_func, 10, avg_time_host, throughput_host, 10);
    LOG(INFO) << "Host vector addition time: " << avg_time_host << " ms";
    LOG(INFO) << "Throughput: " << throughput_host << " GFLOPS" << std::endl;

    // 在设备端分配内存
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // 将数据拷贝到 GPU
    hipMemcpy(d_A, h_A.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), size, hipMemcpyHostToDevice);

    // 设置 CUDA kernel 启动参数
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // 启动 kernel
    auto device_test_func = [&]() {
        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    };
    float device_avg_time, device_throughput;
    util::perf_single_threaded(device_test_func, 10, device_avg_time, device_throughput, 10);
    LOG(INFO) << "Device vector addition time: " << device_avg_time << " ms";
    LOG(INFO) << "Throughput: " << device_throughput << " GFLOPS";
    hipMemcpy(h_C.data(), d_C, size, hipMemcpyDeviceToHost);

    // 验证结果
    util::show_matrix(h_C_host, 1, N, 5);
    util::show_matrix(h_C, 1, N, 5);

    // show time comparison
    LOG(INFO) << "Host time: " << avg_time_host << " ms";
    LOG(INFO) << "Device time: " << device_avg_time << " ms";
    // 释放内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    LOG(INFO) << "Vector addition completed successfully!";
    return 0;
}
