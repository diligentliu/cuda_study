#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <vector>
#include <random>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <glog/logging.h>

#include "cuda_study/util/util.h"
#include "cuda_study/util/perf_util.h"

// C[M,N] = A[M,K] * B[K,N]
__global__ void MatMul(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float value = 0;
        for (int k = 0; k < K; k++) {
            value += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

int main() {
    int M = 50;
    int N = 1024;
    int K = 1024;

    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    std::vector<float> h_A(M * K);
    std::vector<float> h_B(K * N);
    std::vector<float> h_C(M * N);
    std::vector<float> h_C_host(M * N);

    std::mt19937 gen(std::random_device{}());
    std::normal_distribution<float> dist(0.0f, 0.2f);
    for (int i = 0; i < M * K; i++) {
        h_A[i] = dist(gen);
    }
    for (int i = 0; i < K * N; i++) {
        h_B[i] = dist(gen);
    }

    auto host_test_func = [&]() {
        for (int i = 0; i < M; i++) {
            for (int j = 0; j < N; j++) {
                float value = 0;
                for (int k = 0; k < K; k++) {
                    value += h_A[i * K + k] * h_B[k * N + j];
                }
                h_C_host[i * N + j] = value;
            }
        }
    };
    float avg_time, throughput;
    util::perf_single_threaded(host_test_func, 10, avg_time, throughput, 10);
    LOG(INFO) << "Host matrix multiplication time: " << avg_time << " ms";
    LOG(INFO) << "Throughput: " << throughput << " GFLOPS";

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    hipMemcpy(d_A, h_A.data(), size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), size_B, hipMemcpyHostToDevice);

    constexpr int BLOCK_SIZE = 32;
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    auto device_test_func = [&]() {
        MatMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    };
    float device_avg_time, device_throughput;
    util::perf_single_threaded(device_test_func, 10, device_avg_time, device_throughput, 10);
    LOG(INFO) << "Device matrix multiplication time: " << device_avg_time << " ms";
    LOG(INFO) << "Throughput: " << device_throughput << " GFLOPS";

    hipMemcpy(h_C.data(), d_C, size_C, hipMemcpyDeviceToHost);

    if (!util::compare_diff(h_C_host, h_C, M, N, 1e-2f)) {
        LOG(ERROR) << "Result verification failed!";
        goto cleanup;
    }
    LOG(INFO) << "Host time: " << avg_time << " ms";
    LOG(INFO) << "Device time: " << device_avg_time << " ms";

cleanup:
    util::show_matrix(h_C, M, N);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}